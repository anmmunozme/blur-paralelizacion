#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <time.h>
#include <hip/hip_runtime.h>
#define BLOCKSPERGRID 1


//nvcc blur-effect.cu -o blur-effect `pkg-config opencv --libs`

using namespace cv;

//Estructura para separar los 3 canales de color
struct pixel{
 uchar R, G, B;
};

pixel *image;


char add_c[3] = {'b','-',0};

//Calculo de efecto blur recorriendo la imagen y calculando el promedio dado por el kernel
__global__ void blur(pixel *image,int KERNEL_SIZE, int rows, int cols, int numThreads){

	int initIteration;
	int endIteration;
	int i,j, k, l, index;
	index = (blockDim.x * blockIdx.x) + threadIdx.x;
	initIteration = (rows/numThreads)*index;
	endIteration = initIteration + (rows/numThreads) - 1;

    for(i = initIteration; i <= endIteration; i++){
        for(j = 0; j < cols; j++){
                int b = 0, r = 0, g = 0,c = 0;
                for(k = i - KERNEL_SIZE; k <= i + KERNEL_SIZE; k++){
                    for(l = j - KERNEL_SIZE; l <= j + KERNEL_SIZE; l++){
            			b += image[k*cols + l].B;
            			g += image[k*cols + l].G;
            			r += image[k*cols + l].R;
            			c++;
                    }
                }
            	image[i*cols + j].B = (b/c);
            	image[i*cols + j].G = (g/c);
            	image[i*cols + j].R = (r/c);

        }
    }
}



int main(int argc, const char *argv[]){

	Mat im;
	int num_threads;
	int KERNEL_SIZE;
	clock_t tStart = clock();
	hipError_t error = hipSuccess;

	if( argc < 3){
		printf("Usage: <img_path> <kernel_size> <thread_number>\n");
		return -1;
	}


	sscanf(argv[2],"%d", &KERNEL_SIZE);
	sscanf(argv[3],"%d", &num_threads);
	im = imread(argv[1]);

	image = (pixel*)malloc(sizeof(struct pixel)*im.rows*im.cols);


	for(int i = 0; i < im.rows; i++){
		for(int j = 0; j < im.cols; j++){
			image[i*im.cols + j].B = im.at<Vec3b>(i,j)[0];
			image[i*im.cols + j].G = im.at<Vec3b>(i,j)[1];
			image[i*im.cols + j].R = im.at<Vec3b>(i,j)[2];
		}
	}
	pixel *gpu_im;
	error = hipMalloc((void**)&gpu_im,sizeof(struct pixel)*im.rows*im.cols);

	if (error != hipSuccess){
	        fprintf(stderr, "(error allocating in device code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	}

	error = hipMemcpy(gpu_im,image,sizeof(struct pixel)*im.rows*im.cols,hipMemcpyHostToDevice);
	int threadsblocks = (num_threads/BLOCKSPERGRID);


	blur<<<BLOCKSPERGRID,threadsblocks>>>(gpu_im, KERNEL_SIZE, num_threads, im.rows, im.cols);
	error = hipMemcpy(image, gpu_im,sizeof(struct pixel)*im.rows*im.cols,hipMemcpyDeviceToHost);


//cambiar los valores de la imagen por el promedio calculado

	for(int i = 0; i < im.rows; i++){
		for(int j = 0; j < im.cols; j++){
			im.at<Vec3b>(i,j)[0] = image[i*im.cols + j].B;
			im.at<Vec3b>(i,j)[1] = image[i*im.cols + j].G;
			im.at<Vec3b>(i,j)[2] = image[i*im.cols + j].R;
		}
	}

	error = hipFree(gpu_im);

	char * name = (char*)malloc(sizeof(argv[1]) + sizeof(add_c) + 1);
	name[0] = 0;
	strcat(name,add_c);
	strcat(name,argv[1]);
	imwrite(name, im);

	free(image);

	printf("Time taken: %.4fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);

	return 0;
}
